#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows,
                                     int numAColumns, int numBRows,
                                     int numBColumns, int numCRows,
                                     int numCColumns) {
  	//@@ Insert code to implement matrix multiplication here
  	//@@ You have to use shared memory for this MP
	__shared__ float tmpA[16][16];
  	__shared__ float tmpB[16][16];

  	// The element in C a certain thread is in charge of	
  	int C_x = blockIdx.x*blockDim.x + threadIdx.x;
  	int C_y = blockIdx.y*blockDim.y + threadIdx.y;
	float tmp_c_element = 0;

 	// Load tile_idx-th tile from A and B needed to compute current block in C
	for (int tile_idx=0; tile_idx<(numBRows-1)/16+1; ++tile_idx) {
		// Load the element in A and B a certain thread is in charge of
		int A_x = C_x;
		int A_y = tile_idx*blockDim.y + threadIdx.y;
		int B_x = tile_idx*blockDim.x + threadIdx.x;
		int B_y = C_y;
		if (A_x<numARows && A_y<numAColumns)
			tmpA[threadIdx.x][threadIdx.y] = A[A_x*numAColumns+A_y];
		else
			tmpA[threadIdx.x][threadIdx.y] = 0;
		if (B_x<numBRows && B_y<numBColumns)
			tmpB[threadIdx.x][threadIdx.y] = B[B_x*numBColumns+B_y];
		else
			tmpB[threadIdx.x][threadIdx.y] = 0;
		__syncthreads();
		
		// Calculate the element in C a certain thread is in charge of
		if (C_x < numCRows && C_y < numCColumns) {
			for (int i=0; i<16; ++i)
				tmp_c_element += tmpA[threadIdx.x][i]*tmpB[i][threadIdx.y];
		}
		__syncthreads();
	}
	
	if (C_x < numCRows && C_y < numCColumns)
		C[C_x*numCColumns+C_y] = tmp_c_element;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =
      ( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =
      ( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
	
  //@@ Allocate the hostC matrix
  hostC = ( float * )malloc(numCRows*numCColumns*sizeof(float)); 
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**) &deviceA, numARows*numAColumns*sizeof(float));
  hipMalloc((void**) &deviceB, numBRows*numBColumns*sizeof(float));
  hipMalloc((void**) &deviceC, numCRows*numCColumns*sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 grid((numCRows-1)/16+1, (numCColumns-1)/16+1, 1);
  dim3 block(16, 16, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<grid, block>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);	
  wbTime_stop(Copy, "Copying output memory to the CPU");
	
  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}