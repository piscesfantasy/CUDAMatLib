#include <cstdlib>
#include <hip/hip_runtime.h>

/*#include <cuda_runtime_api.h>*/

using namespace std;

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    //@@ Insert code to implement vector addition here
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < len){out[idx]=in1[idx]+in2[idx];}
}

int main(int argc, char **argv) {
    //wbArg_t args;
    int inputLength;
    float *hostInput1;
    float *hostInput2;
    float *hostOutput;
    float *deviceInput1;
    float *deviceInput2;
    float *deviceOutput;

    //args = wbArg_read(argc, argv);

    //hostInput1 = ( float * )wbImport(wbArg_getInputFile(args, 0), &inputLength);
    //hostInput2 = ( float * )wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = ( float * )malloc(inputLength * sizeof(float));

    //@@ Allocate GPU memory here
    hipMalloc((void**) &deviceInput1, inputLength*sizeof(float));
    hipMalloc((void**) &deviceInput2, inputLength*sizeof(float));
    hipMalloc((void**) &deviceOutput, inputLength*sizeof(float));

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput1, hostInput1, inputLength*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength*sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here
    dim3 grid((inputLength-1)/8+1, 1, 1);
    dim3 block(8, 1, 1);

    //@@ Launch the GPU Kernel here
    vecAdd<<<grid, block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(float), hipMemcpyDeviceToHost);	

    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
